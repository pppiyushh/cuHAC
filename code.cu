#include "hip/hip_runtime.h"
#include <math.h>
#include <limits.h>
#include <stdio.h>
#include <cutil_inline.h>
#include <cudpp/cudpp.h>

const int n = 5;
const int d = 2;

//function for calculating distance between two points
__global__ void calculate_distances(float* d_vectors, float* d_distance) {
  int i = blockIdx.x*blockDim.x+threadIdx.x;
  int j = blockIdx.y*blockDim.y+threadIdx.y;

  int index = i + j*n;
  d_distance[index] = 0;

  if (i<n && j<n && i<j) {  
    for (int k=0; k<d; k++) {
      float r = d_vectors[i*d + k] - d_vectors[j*d + k];
      d_distance[index] += r*r;
    }
  }
}
//function for merging the most closet point to the cluster
__global__ void merge_clusters(float* d_distance, int* d_dendrogram, int* d_merged_clusters, int step) {
  int x = blockIdx.x*blockDim.x+threadIdx.x;
  int y = blockIdx.y*blockDim.y+threadIdx.y;
  
  if (x == 0 && y == 0) {
    int min_index = INT_MAX;
    for (int i=0; i<n-1; i++) {
      for (int j=i+1; j<n; j++) {
        if (!d_merged_clusters[i] && !d_merged_clusters[j]) {
          int index = i + j*n;
          if (min_index == INT_MAX || d_distance[index] < d_distance[min_index])
            min_index = index;
        }
      }
    }
    
    if (min_index != INT_MAX) {
      int i = min_index/n;
      int j = min_index%n;
  
      d_dendrogram[step] = i;
      d_dendrogram[step+(n-1)] = j;

      d_merged_clusters[j] = 1;
    }
  }
}

//function which updates the matrix the distances after formation of the clusters 
__global__ void update_distances(float* d_distance, int* d_dendrogram, int* d_merged_clusters, int step) {
  int i = blockIdx.x*blockDim.x+threadIdx.x;
  int j = blockIdx.y*blockDim.y+threadIdx.y;

  int index = i + j*n;

  if (i<n && j<n && i<j) {
    if (!d_merged_clusters[i] && !d_merged_clusters[j]) {
      int candidate_index = INT_MAX;

      if (d_dendrogram[step] == i) candidate_index = d_dendrogram[step+(n-1)] + j*n;
      else if (d_dendrogram[step] == j) candidate_index = d_dendrogram[step+(n-1)] + i*n;
      if (candidate_index != INT_MAX && d_distance[candidate_index] < d_distance[index]) d_distance[index] = d_distance[candidate_index];
    }
  }
}

//function for printing the numerical representation of the dendrogram
void print_step_results(int step, float* h_distance, int* h_dendrogram, int* h_merged_clusters) {
  printf("\n\n\n");    
  printf("Krok %i", step+1);
  printf("\n\n");
  
  printf("Macierz odległości:");
  for (int i=0; i<n; i++) {
    printf("\n");
    for (int j=0; j<n; j++) {
      if (!h_merged_clusters[i] && !h_merged_clusters[j])
        printf(" %f ",h_distance[i*n+j]);
      else
        printf("     M     ");
    }
  }
  printf("\n\n");

  printf("Klastry złączone:");
  printf("\n");
  for (int i=0;i<n;i++) {
    printf(" C%i ",i);
  }
  printf("\n");
  for (int i=0;i<n;i++) {
    printf(" %i ",h_merged_clusters[i]);
  }
  printf("\n\n");

  printf("Dendrogram:");
  for (int i=0;i<(n-1)*2;i++) {
    if (i%(n-1)==0)
      printf("\n");
    printf(" %i ",h_dendrogram[i]);
  }
  printf("\n\n");
}

int main(int argc, char** argv) {
  if (cutCheckCmdLineFlag(argc, (const char**)argv, "device")) cutilDeviceInit(argc, argv);
  else hipSetDevice(cutGetMaxGflopsDeviceId());

  //initializing
  float* h_vectors=(float*)malloc(sizeof(float)* n*d);
  float* h_distance=(float*)malloc(sizeof(float)* n*n);
  int* h_dendrogram=(int*)malloc(sizeof(int)* (n-1)*2);
  int* h_merged_clusters=(int*)malloc(sizeof(int)* n);  

  float* d_vectors;
  cutilSafeCall(hipMalloc((void**)&d_vectors,sizeof(float)* n*d));
  
  float* d_distance;
  cutilSafeCall(hipMalloc((void**)&d_distance,sizeof(float)* n*n));
  
  int* d_dendrogram;
  cutilSafeCall(hipMalloc((void**)&d_dendrogram,sizeof(int)* (n-1)*2));
  
  int* d_merged_clusters;
  cutilSafeCall(hipMalloc((void**)&d_merged_clusters,sizeof(int)* n));

  h_vectors[0*d] = -5;
  h_vectors[0*d+1] = 4;
  
  // P1
  h_vectors[1*d] = 4;
  h_vectors[1*d+1] = -3;
  
  // P2
  h_vectors[2*d] = 5;
  h_vectors[2*d+1] = -5;
  
  // P3
  h_vectors[3*d] = -3;
  h_vectors[3*d+1] = 5;
  
  // P4
  h_vectors[4*d] = 1;
  h_vectors[4*d+1] = 1;

  for (int i=0;i<(n-1)*2;i++) h_dendrogram[i] = 0;
  for (int i=0;i<n;i++) h_merged_clusters[i] = 0;
  
  printf("Punkty wejściowe:");
  for (int i=0;i<n*d;i++) {
    if (i%d==0) {
      printf("\n");
      printf("P%i:", i/d);
    }
    printf(" %f ",h_vectors[i]);
  }
  printf("\n\n\n");
  
  cutilSafeCall(hipMemcpy(d_vectors,h_vectors,n*d*sizeof(float),hipMemcpyHostToDevice));
  
  cutilSafeCall(hipMemcpy(d_dendrogram,h_dendrogram,(n-1)*2*sizeof(int),hipMemcpyHostToDevice));

  cutilSafeCall(hipMemcpy(d_merged_clusters,h_merged_clusters,n*sizeof(int),hipMemcpyHostToDevice));

  int width=n/16+(((n%16)!=0)?1:0);
  int height=n/16+(((n%16)!=0)?1:0);

  dim3 grid(width,height);
  dim3 block(16,16);
  
  dim3 sgrid(1,1);
  dim3 sblock(4,4);

  calculate_distances<<<grid,block>>>(d_vectors, d_distance);
    
  cutilSafeCall(hipMemcpy(h_distance,d_distance,n*n*sizeof(float), hipMemcpyDeviceToHost));
  
  printf("Macierz odległości:");
  for (int i=0;i<n*n;i++) {
    if (i%n==0)
      printf("\n");
    printf(" %f ",h_distance[i]);
  }
  printf("\n\n");
  
  cutilSafeCall(hipMemcpy(d_dendrogram,h_dendrogram,(n-1)*2*sizeof(int),hipMemcpyHostToDevice));
  
  for (int step=0; step < n-1; step++){

    merge_clusters<<<sgrid,sblock>>>(d_distance, d_dendrogram, d_merged_clusters, step);
   
    update_distances<<<grid,block>>>(d_distance, d_dendrogram, d_merged_clusters, step);
    

    cutilSafeCall(hipMemcpy(h_distance,d_distance,n*n*sizeof(float), hipMemcpyDeviceToHost));
    cutilSafeCall(hipMemcpy(h_dendrogram,d_dendrogram,(n-1)*2*sizeof(int), hipMemcpyDeviceToHost));
    cutilSafeCall(hipMemcpy(h_merged_clusters,d_merged_clusters,n*sizeof(int), hipMemcpyDeviceToHost));
    
    print_step_results(step, h_distance, h_dendrogram, h_merged_clusters);
    
  }
  
  hipDeviceReset();
}
